#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>

#include <iostream>

namespace py = pybind11;

// Error Checking Function
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Simple CUDA kernel
__global__ void cuadd(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (index == 0)
        y[0] = x[0] + y[0];
}
