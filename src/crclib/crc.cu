#include <iostream>
#include <sstream>

// #include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include "crc_cpp.hpp"
#include "crc_cuda.cuh"

namespace py = pybind11;

unsigned py_get_crc(py::array_t<unsigned char> vec) {
    py::buffer_info buf = vec.request();
    int len = buf.shape[0];
    unsigned char *ptr = static_cast<unsigned char *>(buf.ptr);
    return get_crc(ptr, len);
}

unsigned py_get_crc_lookup(py::array_t<unsigned char> vec) {
    py::buffer_info buf = vec.request();
    int len = buf.shape[0];
    unsigned char *ptr = static_cast<unsigned char *>(buf.ptr);
    return get_crc_lookup(ptr, len);
}

unsigned py_get_crc_lookup_parallel(py::array_t<unsigned char> vec,
                                    py::array_t<unsigned> table) {
    py::buffer_info vec_buf = vec.request();
    py::buffer_info table_buf = table.request();
    int len = vec_buf.shape[0];
    unsigned char *vec_ptr = static_cast<unsigned char *>(vec_buf.ptr);
    unsigned *table_ptr = static_cast<unsigned *>(table_buf.ptr);
    return get_crc_lookup_parallel(vec_ptr, len, table_ptr);
}

// Simple wrapper function to be exposed to Python
unsigned py_get_crc_cuda(long pycuvec, int len, long py_table, long result) {
    unsigned *d_vec = reinterpret_cast<unsigned *>(pycuvec);
    unsigned *d_table = reinterpret_cast<unsigned *>(py_table);
    unsigned *d_res = reinterpret_cast<unsigned *>(result);
    // Run kernel on 1M elements on the GPU
    int numBlocks = 1;
    int blockSize = 1024;

    crc_cuda<<<numBlocks, blockSize>>>(d_vec, len, d_table, d_res);
    // Wait for GPU to finish before accessing on host
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return 0;
}

PYBIND11_MODULE(mycrclib, m) {
    m.def("get_crc", &py_get_crc);
    m.def("get_crc_lookup", &py_get_crc_lookup);
    m.def("get_crc_lookup_parallel", &py_get_crc_lookup_parallel);
    m.def("get_crc_cuda", &py_get_crc_cuda);
}
